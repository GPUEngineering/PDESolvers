
#include <hip/hip_runtime.h>
#include <iostream>           // Printing
#include <vector>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipblas.h>
#include <memory>             // smart pointers
#include <iomanip>            // for std::setw

#define DEFAULT_FPX double
#if (__cplusplus >= 201703L)  ///< if c++17 or above
#define TEMPLATE_WITH_TYPE_T template<typename T = DEFAULT_FPX>
#else
#define TEMPLATE_WITH_TYPE_T template<typename T>
#endif


//* ================================================================================================
// *  ERROR CHECKING
// * ================================================================================================ */

/**
 * Check for errors when calling GPU functions
 */
#define gpuErrChk(status) { gpuAssert((status), __FILE__, __LINE__); } while(false)

TEMPLATE_WITH_TYPE_T inline void gpuAssert(T code, const char *file, int line, bool abort = true) {
    if constexpr (std::is_same_v<T, hipError_t>) {
        if (code != hipSuccess) {
            std::cerr << "cuda error. String: " << hipGetErrorString(code)
                      << ", file: " << file << ", line: " << line << "\n";
            if (abort) exit(code);
        }
    } else if constexpr (std::is_same_v<T, hipblasStatus_t>) {
        if (code != HIPBLAS_STATUS_SUCCESS) {
            std::cerr << "cublas error. Name: " << cublasGetStatusName(code)
                      << ", string: " << cublasGetStatusString(code)
                      << ", file: " << file << ", line: " << line << "\n";
            if (abort) exit(code);
        }
    } else if constexpr(std::is_same_v<T, hipsparseStatus_t>) {
        if (code != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "cublas error. Code: " << hipsparseGetErrorString(code)
                      << ", file: " << file << ", line: " << line << "\n";
            if (abort) exit(code);
        }
    } else {
        std::cerr << "Error: library status parser not implemented" << "\n";
    }
}



/* ================================================================================================
 *  SESSION
 * ================================================================================================ */
/**
 * Singleton for Cuda library handles
 */
class Session {
public:

    static Session &getInstance() {
        static Session instance;
        return instance;
    }

private:
    Session() {
        gpuErrChk(hipblasCreate(&m_cublasHandle));
        gpuErrChk(hipsparseCreate(&m_sparseHandle));
    }

    ~Session() {
        gpuErrChk(hipblasDestroy(m_cublasHandle));
        gpuErrChk(hipsparseDestroy(m_sparseHandle));
    }

    hipblasHandle_t m_cublasHandle;
    hipsparseHandle_t m_sparseHandle;


public:
    Session(Session const &) = delete;

    void operator=(Session const &) = delete;

    hipblasHandle_t &cuBlasHandle() { return m_cublasHandle; }

    hipsparseHandle_t &cuSpraseHandle() { return m_sparseHandle; }
};


/* ================================================================================================
 *  DVector
 * ================================================================================================ */
TEMPLATE_WITH_TYPE_T
class DVector {
private:
    size_t m_nulEl = 0;
    T *m_d_data = nullptr;
    hipsparseDnVecDescr_t m_vecX;

public:

    DVector(size_t n) {
        m_nulEl = n;
        gpuErrChk(hipMalloc((void **) &m_d_data, m_nulEl * sizeof(T)));
        gpuErrChk(hipsparseCreateDnVec(&m_vecX, m_nulEl, m_d_data, HIP_R_32F));
    }

    DVector(std::vector<T> hostData) {
        m_nulEl = hostData.size();
        gpuErrChk(hipMalloc((void **) &m_d_data, m_nulEl * sizeof(T)));
        gpuErrChk(hipMemcpy(m_d_data, hostData.data(), m_nulEl * sizeof(T), hipMemcpyHostToDevice));
        gpuErrChk(hipsparseCreateDnVec(&m_vecX, m_nulEl, m_d_data, HIP_R_32F));
    }

    ~DVector() {
        if (m_d_data) {
            gpuErrChk(hipFree(m_d_data));
            m_d_data = nullptr;
            gpuErrChk(hipsparseDestroyDnVec(m_vecX));
        }
        m_nulEl = 0;
    }

    hipsparseDnVecDescr_t &asCusparseVector() {
        return m_vecX;
    }

    void downloadTo(T *hostData) {
        gpuErrChk(hipMemcpy(hostData, m_d_data, m_nulEl * sizeof(T), hipMemcpyDeviceToHost));
    }

    void downloadTo(std::vector<T> &vec) const {
        vec.resize(m_nulEl);
        gpuErrChk(hipMemcpy(vec.data(),
                             m_d_data,
                             m_nulEl * sizeof(T),
                             hipMemcpyDeviceToHost));
    }

    size_t numEl() {
        return m_nulEl;
    }

    void deviceCopyFrom(DVector<T> &other) {
        gpuErrChk(hipMemcpy(m_d_data, other.m_d_data, m_nulEl * sizeof(T), hipMemcpyDeviceToDevice));
    }

    std::ostream &print(std::ostream &out) const {
        std::vector<T> temp;
        downloadTo(temp);
        out << "[DVector] " << m_nulEl << " elements " << std::endl;
        for (size_t i = 0; i < m_nulEl; i++) {
            out << std::setw(10) << temp[i] << ", " << std::endl;
        }
        return out;
    }

    friend std::ostream &operator<<(std::ostream &out, const DVector<T> &data) {
        return data.print(out);
    }

    T norm() const;

};

template<>
inline double DVector<double>::norm() const {
    double the_norm;
    gpuErrChk(hipblasDnrm2(Session::getInstance().cuBlasHandle(), m_nulEl, m_d_data, 1, &the_norm));
    return the_norm;
}

template<>
inline float DVector<float>::norm() const {
    float the_norm;
    gpuErrChk(hipblasSnrm2(Session::getInstance().cuBlasHandle(), m_nulEl, m_d_data, 1, &the_norm));
    return the_norm;
}

/* ================================================================================================
 *  DSparseCSRMatrix (CSR SPARSE MATRIX)
 * ================================================================================================ */
TEMPLATE_WITH_TYPE_T
class DSparseCSRMatrix {
private:
    /* Metadata */
    size_t m_numRows = 0;  ///< Number of rows
    size_t m_numCols = 0;  ///< Number of columns
    size_t m_numNonZeros = 0;  ///< Number of nonzero elements

    /* Data */
    T *m_d_data = nullptr;  ///< Pointer to device data
    int *m_d_csrOffsets = nullptr;
    int *m_d_csrColumns = nullptr;
    hipsparseSpMatDescr_t m_csrMat;

    /* Buffer */
    size_t m_bufferSize = 0;
    void *m_buffer = nullptr;

public:
    DSparseCSRMatrix(const std::vector<T> &data,
                     const std::vector<int> &csrOffsets,
                     const std::vector<int> &csrColumns,
                     size_t nRows,
                     size_t nCols,
                     size_t nNonzero) :
            m_numCols(nCols), m_numRows(nRows), m_numNonZeros(nNonzero) {
        /* allocate memory */
        gpuErrChk(hipMalloc((void **) &m_d_data, m_numNonZeros * sizeof(T)));
        gpuErrChk(hipMalloc((void **) &m_d_csrOffsets, (m_numRows + 1) * sizeof(int)));
        gpuErrChk(hipMalloc((void **) &m_d_csrColumns, m_numNonZeros * sizeof(int)));
        /* copy data to device */
        gpuErrChk(hipMemcpy(m_d_data, data.data(), m_numNonZeros * sizeof(T), hipMemcpyHostToDevice));
        gpuErrChk(hipMemcpy(m_d_csrOffsets, csrOffsets.data(), (m_numRows + 1) * sizeof(int), hipMemcpyHostToDevice));
        gpuErrChk(hipMemcpy(m_d_csrColumns, csrColumns.data(), m_numNonZeros * sizeof(int), hipMemcpyHostToDevice));
        /* create CSR */
        gpuErrChk(hipsparseCreateCsr(&m_csrMat, m_numRows, m_numCols, m_numNonZeros,
                                    m_d_csrOffsets, m_d_csrColumns, m_d_data,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    }

    ~DSparseCSRMatrix() {
        if (m_d_csrOffsets) {
            gpuErrChk(hipFree(m_d_csrOffsets));
            m_d_csrOffsets = nullptr;
        }
        if (m_d_csrColumns) {
            gpuErrChk(hipFree(m_d_csrColumns));
            m_d_csrColumns = nullptr;
        }
        if (m_d_data) {
            gpuErrChk(hipFree(m_d_data));
            m_d_data = nullptr;
        }
        if (m_buffer) {
            gpuErrChk(hipFree(m_buffer));
            m_buffer = nullptr;
        }
        if (m_numNonZeros) gpuErrChk(hipsparseDestroySpMat(m_csrMat));
    }

    /**
     * Performs y = alpha * A * x + beta * y
     * @param y vector
     * @param x vector
     * @param alpha scalar
     * @param beta scalar
     */
    void axpby(DVector<T> &y,
               DVector<T> &x,
               T alpha = 1.,
               T beta = 0) {
        if (!m_buffer) {
            gpuErrChk(hipsparseSpMV_bufferSize(
                    Session::getInstance().cuSpraseHandle(),
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha, m_csrMat, x.asCusparseVector(), &beta, y.asCusparseVector(), HIP_R_32F,
                    HIPSPARSE_SPMV_ALG_DEFAULT, &m_bufferSize));
            gpuErrChk(hipMalloc((void **) &m_buffer, m_bufferSize));
        }
        gpuErrChk(hipsparseSpMV(Session::getInstance().cuSpraseHandle(),
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, m_csrMat, x.asCusparseVector(), &beta, y.asCusparseVector(), HIP_R_32F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, m_buffer));
    }

    size_t nRows() {
        return m_numRows;
    }

    size_t nCols() {
        return m_numCols;
    }

};


TEMPLATE_WITH_TYPE_T
class CGSolver {
private:
    DSparseCSRMatrix<T> &m_lhs;
    std::unique_ptr<DVector<T>> m_residual = nullptr;
    std::unique_ptr<DVector<T>> m_search_direction = nullptr;
public:
    CGSolver(DSparseCSRMatrix<T> &lhsMatrix) : m_lhs(lhsMatrix) {
        size_t m = m_lhs.nRows();
        m_residual = std::make_unique<DVector<T>>(m);
        m_search_direction = std::make_unique<DVector<T>>(m);
    }

    void solve(DVector<T> &rhs, DVector<T> &x, T eps) {
        // We want to do r = b - Ax, i.e,.
        m_residual->deviceCopyFrom(rhs); // 1. r = b
        m_lhs.axpby(*m_residual, rhs, -1, 1);// 2. r = -1Ax + 1r

        m_search_direction->deviceCopyFrom(*m_residual);
        T norm_x = x.norm();
        T old_resid_norm = m_residual->norm();
        size_t max_iter = m_lhs.nCols();
        for (size_t i = 0; i < max_iter; i++) {

        }
    }

};



//* ================================================================================================
// *  MAIN function (for testing only)
// * ================================================================================================ */

int main(void) {

    // MATRIX A DATA (CSR)
    const int nr = 4;
    const int nc = 4;
    const int nnz = 9;
    DSparseCSRMatrix<float> aCSR(std::vector<float>{1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f},
                                 std::vector<int>{0, 3, 4, 7, 9},
                                 std::vector<int>{0, 2, 3, 1, 0, 2, 3, 1, 3},
                                 nr, nc, nnz);

    // VECTORS
    DVector<float> x(std::vector<float>{1., 2., 3., 4.});
    DVector<float> b(std::vector<float>{38., 16., 102., 104.});
    std::cout << b;
    std::cout << x;

    CGSolver<float> solver(aCSR);
    solver.solve(b, x, 0.01);

    std::cout << b;
    std::cout << x;

    return EXIT_SUCCESS;
}