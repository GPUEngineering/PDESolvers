//
// Created by Chelsea De Marseilla on 03/01/2025.
//


#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

#define SEED 12345

class GBMParallel
{
    private:
        float initial_stock_price;
        float mu;
        float sigma;
        float time;
        int time_steps;
        int num_of_simulations;

    public:
        GBMParallel(float initial_stock_price, float mu, float sigma, float time, int time_steps, int num_of_simulations)
        {
            this->initial_stock_price = initial_stock_price;
            this->mu = mu;
            this->sigma = sigma;
            this->time = time;
            this->time_steps = time_steps;
            this->num_of_simulations = num_of_simulations;
        }

        // sets cuda kernel to run on gpu
        __global__ static void simulate_gbm(float* grid, float* brownian_path, float initial_stock_price, float mu, float sigma, float time, int time_steps, int num_of_simulations)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;

            if (idx < num_of_simulations)
            {

                // initialising for random normal distribution
                hiprandState state;
                hiprand_init(SEED, idx, 0, &state);

                float dt = time/static_cast<float>(time_steps);

                brownian_path[idx * time_steps] = 0.0f;
                grid[idx * time_steps] = initial_stock_price;

                for (int i = 1; i < time_steps; i++)
                {
                    float Z = hiprand_normal(&state);
                    brownian_path[idx * time_steps + i] = brownian_path[idx * time_steps + i - 1] + std::sqrt(dt) * Z;
                    grid[idx * time_steps + i] = grid[idx * time_steps + i - 1] * expf((mu - 0.5f * powf(sigma, 2)) * dt + sigma * (brownian_path[idx * time_steps + i] - brownian_path[idx * time_steps + i - 1]));
                }
            }
        }


};

int main()
{

    float initial_stock_price = 100.0f;
    float mu = 0.05f;
    float sigma = 0.03f;
    float time = 1;
    int time_steps = 365;
    int num_of_simulations = 100;

    int block_size = 256;
    int num_blocks = (num_of_simulations + block_size - 1) / block_size;

    size_t grid_size = num_of_simulations * time_steps;

    // host memory allocation
    float* host_grid = (float*)malloc(grid_size * sizeof(float));

    // gpu memory allocation
    float *dev_grid, *bm;
    hipMalloc(&dev_grid, grid_size * sizeof(float));
    hipMalloc(&bm, num_of_simulations * time_steps * sizeof(float));

    // copy host memory to gpu memory
    hipMemcpy(dev_grid, host_grid, grid_size * sizeof(float), hipMemcpyHostToDevice);

    // kernel invocation
    GBMParallel::simulate_gbm<<<num_blocks, block_size>>>(dev_grid, bm, initial_stock_price, mu, sigma, time, time_steps, num_of_simulations);

    // waits kernel to finish all processes
    hipDeviceSynchronize();

    // copy updated gpu memory to host memory
    hipMemcpy(host_grid, dev_grid,grid_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_grid);
    hipFree(bm);

    free(host_grid);
}