//
// Created by Chelsea De Marseilla on 03/01/2025.
//


#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

#define SEED 12345

__global__ static void simulate_gbm(float* grid, float* brownian_path, float initial_stock_price, float mu, float sigma, float time, int time_steps, int num_of_simulations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_of_simulations)
    {

        // initialising for random normal distribution
        hiprandState state;
        hiprand_init(SEED, idx, 0, &state);

        float dt = time/static_cast<float>(time_steps);

        brownian_path[idx * time_steps] = 0.0f;
        grid[idx * time_steps] = initial_stock_price;

        for (int i = 1; i < time_steps; i++)
        {
            float Z = hiprand_normal(&state);
            brownian_path[idx * time_steps + i] = brownian_path[idx * time_steps + i - 1] + std::sqrt(dt) * Z;
            grid[idx * time_steps + i] = grid[idx * time_steps + i - 1] * expf((mu - 0.5f * powf(sigma, 2)) * dt + sigma * (brownian_path[idx * time_steps + i] - brownian_path[idx * time_steps + i - 1]));
        }
    }
}

int main()
{

    float initial_stock_price = 100.0f;
    float mu = 0.05f;
    float sigma = 0.03f;
    float time = 1;
    int time_steps = 365;
    int num_of_simulations = 100;

    int block_size = 256;
    int num_blocks = (num_of_simulations + block_size - 1) / block_size;

    size_t grid_size = num_of_simulations * time_steps;

    // host memory allocation
    float* host_grid = (float*)malloc(grid_size * sizeof(float));

    // gpu memory allocation
    float *dev_grid, *bm;
    hipMalloc(&dev_grid, grid_size * sizeof(float));
    hipMalloc(&bm, num_of_simulations * time_steps * sizeof(float));

    // copy host memory to gpu memory
    hipMemcpy(dev_grid, host_grid, grid_size * sizeof(float), hipMemcpyHostToDevice);

    // kernel invocation
    simulate_gbm<<<num_blocks, block_size>>>(dev_grid, bm, initial_stock_price, mu, sigma, time, time_steps, num_of_simulations);

    // waits kernel to finish all processes
    hipDeviceSynchronize();

    // copy updated gpu memory to host memory
    hipMemcpy(host_grid, dev_grid,grid_size * sizeof(float), hipMemcpyDeviceToHost);


    // printing for debugging
    for (int i = 0; i < 10; i++)
    {
        std::cout << "Simulation " << i << ": ";
        for (int j = 0; j < time_steps; j++)
        {
            std::cout << host_grid[i * time_steps + j] << " ";
        }
        std::cout << std::endl;
    }

    hipFree(dev_grid);
    hipFree(bm);

    free(host_grid);
}
